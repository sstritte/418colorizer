#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <float.h>
#include <utility>
#include <stdio.h>
#include <cstring>
#include <vector>
#include <unistd.h>
#include "cudaRenderer.h"
#include "image.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#define CELL_DIM 1
#define TIME_STEP 1


///////////////////////////CUDA CODE BELOW////////////////////////////////
struct GlobalConstants {
    int cells_per_side;
    int width;
    int height;

    float* VX;
    float* VY;
    float* pressures;
    float* VXCopy;
    float* VYCopy;
    float* divergence;
    float* vorticity;
    float* color;
    float* colorCopy;
    float* imageData;

    int* mpls;
};

__constant__ GlobalConstants cuParams;

// kernelClearImage
__global__ void kernelClearImage(float r, float g, float b, float a) {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuParams.width;
    int height = cuParams.height;

    if (imageX >= width || imageY >= height) return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r,g,b,a);
    
    // Write to global memory.
    *(float4*)(&cuParams.imageData[offset]) = value;

}


__device__ __inline__ int
isBoundary(int i, int j) {
    int cells_per_side = cuParams.cells_per_side;
    if (j == 0) return 1; // left 
    if (i == 0) return 2; // top
    if (j == cells_per_side) return 3; // right
    if (i == cells_per_side) return 4; // bottom
    return 0;
}

// a is prev mouse point, b is cur mouse point, p is point to consider,
// fp is fraction projection to be populated as output
__device__ __inline__ double 
distanceToSegment(double ax, double ay, double bx, double by, 
        double px, double py, double* fp) {
    double dx = px - ax; //vec2 d = p - a;
    double dy = py - ay;
    double xx = bx - ax; //vec2 x = b - a;
    double xy = by - ay;
    *fp = 0.0; // fractional projection, 0 - 1 in the length of b-a
    double lx = sqrt(xx*xx + xy*xy); //length(x)
    double ld = sqrt(dx*dx + dy*dy); //length(d)
    if (lx <= 0.0001) return ld;
    double projection = dx*(xx/lx) + dy*(xy/lx); //dot(d, x/lx)
    *fp = projection / lx;
    if (projection < 0.0) return ld;
    else if (projection > lx) return sqrt((px-bx) * (px-bx) +
            (py-by) * (py-by)); //length(p - b)
    return sqrt(abs(dx*dx + dy*dy - projection * projection));
}

__device__ __inline__ double 
distanceToNearestMouseSegment(double px, double py, double *fp,
        double* vx, double *vy) {
    double minLen = DBL_MAX;
    double fpResult = 0.0;
    double vxResult = 0.0;
    double vyResult = 0.0;
    for (int i = 0; i < 400 - 2; i += 2) {

        int grid_col1 = cuParams.mpls[i];
        int grid_row1 = cuParams.mpls[i + 1];
        int grid_col2 = cuParams.mpls[i + 2];
        int grid_row2 = cuParams.mpls[i + 3];
        if (grid_col2 == 0 & grid_row2 == 0) break;
        double len = distanceToSegment(grid_col1, grid_row1, grid_col2, grid_row2, px, py, fp);
        if (len < minLen) {
            minLen = len;
            fpResult = *fp;
            vxResult = grid_col2 - grid_col1;
            vyResult = grid_row2 - grid_row1;
        }        

    }
    *fp = fpResult;
    *vx = vxResult;
    *vy = vyResult;
    return minLen;
}

//kernelFadeVelocities
__global__ void kernelFadeVelocities() {
    int grid_col = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;
    int height = cuParams.height;

    if (grid_col >= width || grid_row >= height) return;
    if (grid_row * width + grid_col >= width * height) return; 
    
    cuParams.VX[grid_row * width + grid_col] *= 0.999;
    cuParams.VY[grid_row * width + grid_col] *= 0.999;
}

//kernelSetNewVelocities
__global__ void kernelSetNewVelocities() {
    int grid_col = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;
    int height = cuParams.height;

    if (grid_col >= width || grid_row >= height) return;
    if (grid_row * width + grid_col >= width * height) return; 
    
    int imageX = grid_col;
    int imageY = grid_row;
    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(1.f,0.f,1.f,1.f);

    // Write to global memory.
    *(float4*)(&cuParams.imageData[offset]) = value;
   
    cuParams.VX[grid_row * width + grid_col] *= 0.999;
    cuParams.VY[grid_row * width + grid_col] *= 0.999;
    double projection;
    double vx;
    double vy;
    double l = distanceToNearestMouseSegment(grid_col, grid_row, 
            &projection, &vx, &vy);
    //printf("velocity %f,%f\n", mouseSegmentVelocity.first, mouseSegmentVelocity.second);
    double taperFactor = 0.6;
    double projectedFraction = 1.0 - fminf(1.0, fmaxf(projection, 0.0)) * taperFactor;
    double R = 10;
    double m = exp(-l/R); //drag coefficient
    m *= projectedFraction * projectedFraction;
    double targetVelocityX = vx * 1 * 1.4; 
    double targetVelocityY = vy * 1 * 1.4; 

    cuParams.VX[grid_row * width + grid_col] += 
        (targetVelocityX - cuParams.VX[grid_row * width + grid_col]) * m;
    cuParams.VY[grid_row * width + grid_col] += 
        (targetVelocityY - cuParams.VY[grid_row * width + grid_col]) * m;

}

//kernelAdvectVelocityForward
__global__ void kernelAdvectVelocityForward() {
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    cuParams.VXCopy[row * width + col] = cuParams.VX[row * width + col];
    cuParams.VYCopy[row * width + col] = cuParams.VY[row * width + col];

   int pixelRow = row * CELL_DIM;
   int pixelCol = col * CELL_DIM;
   int nextPixelRow = round(pixelRow + TIME_STEP * cuParams.VY[row * width + col] * CELL_DIM);
   int nextPixelCol = round(pixelCol + TIME_STEP * cuParams.VX[row * width + col] * CELL_DIM);
   int nextCellCol = nextPixelCol / CELL_DIM;
   int nextCellRow = nextPixelRow / CELL_DIM;

   if (nextCellCol < cells_per_side && nextCellRow < cells_per_side 
           && nextCellCol >= 0 && nextCellRow >= 0) {
        cuParams.VX[nextCellRow * width + nextCellCol] = cuParams.VXCopy[row * width + col];
        cuParams.VY[nextCellRow * width + nextCellCol] = cuParams.VYCopy[row* width + col];
   } 

}

//kernelAdvectVelocityBackward
__global__ void kernelAdvectVelocityBackward() {
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

   int pixelRow = row * CELL_DIM;
   int pixelCol = col * CELL_DIM;
   int prevPixelRow = round(pixelRow - TIME_STEP * cuParams.VY[row * width + col] * CELL_DIM);
   int prevPixelCol = round(pixelCol - TIME_STEP * cuParams.VX[row * width + col] * CELL_DIM);
   int prevCellCol = prevPixelCol / CELL_DIM;
   int prevCellRow = prevPixelRow / CELL_DIM;

   if (prevCellCol < cells_per_side && prevCellRow < cells_per_side 
           && prevCellCol >= 0 && prevCellRow >= 0) {
        cuParams.VX[row * width + col] = cuParams.VXCopy[prevCellRow * width + prevCellCol];
        cuParams.VY[row * width + col] = cuParams.VYCopy[prevCellRow * width + prevCellCol];
   } 
   if (prevCellCol == col && prevCellRow == row) {
        // you don't move so just disappear
        cuParams.VX[row * width + col] = 0;
        cuParams.VY[row * width + col] = 0;
   }
}

//kernelApplyVorticity
__global__ void kernelApplyVorticity(){
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    if (isBoundary(row,col)) return;

    float L = 0.0;
    float R = 0.0;
    float B = 0.0;
    float T = 0.0;

    if (row > 0) T = cuParams.VX[(row-1) * width + col];
    if (row < cells_per_side) B = cuParams.VX[(row+1) * width + col];
    if (col < cells_per_side) R = cuParams.VY[row * width + (col+1)];
    if (col > 0) L = cuParams.VY[row * width + (col-1)];
    cuParams.vorticity[row * width + col] = 0.5 * ((R - L) - (T - B));
}

//kernelApplyVorticityForce
__global__ void kernelApplyVorticityForce(){
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    float vortConfinementFloat = 0.035f;
    float vortL = 0.0;
    float vortR = 0.0;
    float vortB = 0.0;
    float vortT = 0.0;
    float vortC = 0.0;
            
    if (isBoundary(row,col)) return;

    if (row > 0) vortT = cuParams.vorticity[(row-1) * width + col];
    if (row < cells_per_side) vortB = cuParams.vorticity[(row+1) * width + col];
    if (col < cells_per_side) vortR = cuParams.vorticity[row * width + (col+1)];
    if (row > 0) vortL = cuParams.vorticity[row * width + (col-1)];
    vortC = cuParams.vorticity[row * width + col];
    
    float forceX = 0.5 * (fabsf(vortT) - fabsf(vortB));
    float forceY = 0.5 * (fabsf(vortR) - fabsf(vortL));
    float EPSILON = powf(2,-12);
    float magSqr = fmaxf(EPSILON, forceX * forceX + forceY * forceY);
    forceX = forceX * (1/sqrtf(magSqr));
    forceY = forceY * (1/sqrtf(magSqr));
    forceX *= vortConfinementFloat * vortC * 1;
    forceY *= vortConfinementFloat * vortC * -1;
    cuParams.VX[row * width + col] += forceX;
    cuParams.VY[row * width + col] += forceY;
}

//kernelApplyDivergence
__global__ void kernelApplyDivergence(){
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    if (isBoundary(row,col)) return;

    float L = 0.0;
    float R = 0.0;
    float B = 0.0;
    float T = 0.0;

    if (row > 0) T = cuParams.VY[(row-1) * width + col];
    if (row < cells_per_side) B = cuParams.VY[(row+1) * width + col];
    if (col < cells_per_side) R = cuParams.VX[row * width + (col+1)];
    if (col > 0) L = cuParams.VX[row * width + (col-1)];
    cuParams.divergence[row * width + col] = 0.5*((R-L) + (T-B));
}

//kernelPressureSolve
__global__ void kernelPressureSolve(){
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    if (isBoundary(row,col)) return;

    float L = 0.0;
    float R = 0.0;
    float B = 0.0;
    float T = 0.0;

    if (row > 0) T = cuParams.pressures[(row-1) * width + col];
    if (row < cells_per_side) B = cuParams.pressures[(row+1) * width + col];
    if (col < cells_per_side) R = cuParams.pressures[row * width + (col+1)];
    if (col > 0) L = cuParams.pressures[row * width + (col-1)];
    // I FEEL LIKE MAYBE WE NEED A SYNCTHREADS() HERE!!!!!!!
    // BECAUSE IN THE REF WE HAVE A TEMPPRESSURES ARRAY AND COPY EVERYTHING AT THE END
    // SO THAT EVERYONE IS ALWAYS READING FROM THE OLD PRESSURE ARRAY AND THINGS AREN'T BEING
    // UPDATED AS THEY GO
    // ALTERNATIVELY WE COULD HAVE THIS KERNEL MAKE A cuParams.tempPressures IN THIS WAY
    // AND THEN HAVE  SEPARATE KERNEL (TO CALL AFTER THIS ONE) THAT COPIES cuParams.tempPressures
    // OVER TO the real cuParams.pressures
    cuParams.pressures[row * width + col] = (L + R + B + T + -1 * cuParams.divergence[row * width + col]) * .25;
}

//kernelPressureGradient
__global__ void kernelPressureGradient(){
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    if (isBoundary(row,col)) return;

    float L = 0.0;
    float R = 0.0;
    float B = 0.0;
    float T = 0.0;

    if (row > 0) T = cuParams.pressures[(row-1) * width + col];
    if (row < cells_per_side) B = cuParams.pressures[(row+1) * width + col];
    if (col < cells_per_side) R = cuParams.pressures[row * width + (col+1)];
    if (col > 0) L = cuParams.pressures[row * width + (col-1)];
    cuParams.VX[row * width + col] = cuParams.VX[row * width + col] - 0.5*(R - L);
    cuParams.VY[row * width + col] = cuParams.VY[row * width + col] - 0.5*(T - B);
}

//kernelAdvectColorForward
__global__ void kernelAdvectColorForward() {
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    // I HAVE A SIMILAR CONCERN HERE AS THE COMMENT I WROTE ABOVE...
    // DO WE NEED ALL THE KERNELS TO FINISH POPULATING cuParams.colorCopy
    // BEFORE THEY START THE REST OF THE LOGIC??? SINCE EACH ACCESSES OTHER PLACES IN
    // cuParams.colorCopy BELOW. (THIS SAME QUESTION WILL HAVE TO APPLY TO 
    // kernelAdvectVelocityForward()...)
    cuParams.colorCopy[row * width + col] = cuParams.color[row * width + col];

    int pixelRow = row * CELL_DIM;
    int pixelCol = col * CELL_DIM;
    // THERE ARE A ZILLION VERSION OF ROUNDING FUNCTIONS IN THE CUDA MATH API
    // http://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__SINGLE.html#group__CUDA__MATH__SINGLE
    // I THINK WE WANT TO CHANGE round(x) TO rint(x).... BUT NOT SURE
    // SAME CHANGE WOULD NEED TO BE APPLIED IN kernelAdvectVelocityForward() and kernelAdvectVelocityBackward()
    // and kernelAdvectColorBackward()
    int nextPixelRow = round(pixelRow + TIME_STEP * cuParams.VY[row * width + col] * CELL_DIM);
    int nextPixelCol = round(pixelCol + TIME_STEP * cuParams.VX[row * width + col] * CELL_DIM);
    int nextCellCol = nextPixelCol / CELL_DIM;
    int nextCellRow = nextPixelRow / CELL_DIM;

   if (nextCellCol < cells_per_side && nextCellRow < cells_per_side 
           && nextCellCol >= 0 && nextCellRow >= 0) {
        // I SPENT LIKE 10 MINUTES TRYING TO CONVINCE MYSELF IF THE col * 4 THING IS RIGHT 
        // BUT I'M STILL NOT CONVINCED SO WE SHOULD DOUBLE/TRIPLE CHECK THIS
        cuParams.color[nextCellRow * width + nextCellCol * 4 + 0] = cuParams.colorCopy[row * width + col * 4 + 0];
        cuParams.color[nextCellRow * width + nextCellCol * 4 + 1] = cuParams.colorCopy[row * width + col * 4 + 1];
        cuParams.color[nextCellRow * width + nextCellCol * 4 + 2] = cuParams.colorCopy[row * width + col * 4 + 2];
        cuParams.color[nextCellRow * width + nextCellCol * 4 + 3] = cuParams.colorCopy[row * width + col * 4 + 3];

   } 
}

//kernelAdvectColorBackward
__global__ void kernelAdvectColorBackward() {
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    int pixelRow = row * CELL_DIM;
    int pixelCol = col * CELL_DIM;
    int prevPixelRow = round(pixelRow - TIME_STEP * cuParams.VY[row * width + col] * CELL_DIM);
    int prevPixelCol = round(pixelCol - TIME_STEP * cuParams.VX[row * width + col] * CELL_DIM);
    int prevCellCol = prevPixelCol / CELL_DIM;
    int prevCellRow = prevPixelRow / CELL_DIM;

    if (prevCellCol < cells_per_side && prevCellRow < cells_per_side 
            && prevCellCol >= 0 && prevCellRow >= 0) {
         cuParams.color[row * width + col * 4 + 0] = cuParams.colorCopy[prevCellRow * width + prevCellCol * 4 + 0];
         cuParams.color[row * width + col * 4 + 1] = cuParams.colorCopy[prevCellRow * width + prevCellCol * 4 + 1];
         cuParams.color[row * width + col * 4 + 2] = cuParams.colorCopy[prevCellRow * width + prevCellCol * 4 + 2];
         cuParams.color[row * width + col * 4 + 3] = cuParams.colorCopy[prevCellRow * width + prevCellCol * 4 + 3];
   } 
}


//////////////////////////////////////////////////////////////////////////
///////////////////////////HOST CODE BELOW////////////////////////////////
//////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
    image = NULL;

    VX = NULL;
    VY = NULL;
    color = NULL;
    colorCopy = NULL;
    pressures = NULL;
    VXCopy = NULL;
    VYCopy = NULL;
    divergence = NULL;
    vorticity = NULL;

    mpls = NULL;

    cdVX = NULL;
    cdVY = NULL;
    cdColor = NULL;
    cdColorCopy = NULL;
    cdPressures = NULL;
    cdVXCopy = NULL;
    cdVYCopy = NULL;
    cdDivergence = NULL;
    cdVorticity = NULL;
    cdImageData = NULL;

    cdMpls = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) delete image;

    if (VX) {
        delete VX;
        delete VY;
        delete pressures;
        delete VXCopy;
        delete VYCopy;
        delete divergence;
        delete vorticity;
        delete color;
        delete colorCopy;
        delete mpls;
    }

    if (cdVX) {
        hipFree(cdVX);
        hipFree(cdVY);
        hipFree(cdPressures);
        hipFree(cdVXCopy);
        hipFree(cdVYCopy);
        hipFree(cdDivergence);
        hipFree(cdVorticity);
        hipFree(cdColor);
        hipFree(cdColorCopy);
        hipFree(cdImageData);
        hipFree(cdMpls);
    }
}

const Image*
CudaRenderer::getImage() {
    printf("Copying image data from device\n");

    hipMemcpy(image->data, cdImageData, 
            4 * sizeof(float) * image->width * image->height,
            hipMemcpyDeviceToHost);

    return image;
}


void
CudaRenderer::setup() {
   cells_per_side = image->width / CELL_DIM - 1;

   hipMalloc(&cdVX, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdVY, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdPressures, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdVXCopy, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdVYCopy, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdDivergence, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdVorticity, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdColor, 4 * sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdColorCopy, 4 * sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdImageData, 4 * sizeof(float) * image->width * image->height);
   hipMalloc(&cdMpls, 400 * sizeof(float) * image->width * image->height);

   hipMemset(cdVX, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdVY, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdPressures, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdVXCopy, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdVYCopy, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdDivergence, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdVorticity, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdColor, 0, 4 * sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdColorCopy, 0, 4 * sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));

    GlobalConstants params;
    params.cells_per_side = cells_per_side;
    params.width = image->width;
    params.height = image->height;
    params.VX = cdVX;
    params.VY = cdVY;
    params.pressures = cdPressures;
    params.VXCopy = cdVXCopy;
    params.VYCopy = cdVYCopy;
    params.divergence = cdDivergence;
    params.vorticity = cdVorticity;
    params.color = cdColor;
    params.colorCopy = cdColorCopy;
    params.imageData = cdImageData;
    params.mpls = cdMpls;

    hipMemcpyToSymbol(HIP_SYMBOL(cuParams), &params, sizeof(GlobalConstants));
}

// Called after clear, before render
void CudaRenderer::setNewQuantities(std::vector<std::pair<int, int> > mpls) {

    int mplsSize = mpls.size();
    if (mplsSize == 0) {
        // if mpls.size is 0, then call kernel that decreases VX,VY by 0.999
        dim3 blockDim(16,16,1);
        dim3 gridDim(
                (image->width + blockDim.x - 1) / blockDim.x,
                (image->height + blockDim.y - 1) / blockDim.y);
        kernelFadeVelocities<<<gridDim, blockDim>>>();
        hipDeviceSynchronize();

    } else {
        int* mplsArray = new int[mplsSize * 2 + 1];
        int count = 0;
        for (std::vector<std::pair<int,int> >::iterator it = mpls.begin() 
                ; it != mpls.end(); ++it) {
            std::pair<int,int> c = *it;
            mplsArray[count] = c.first;
            mplsArray[count + 1] = c.second;
            count += 2;
        }
        hipMemset(cdMpls, 0, 400 * sizeof(int));
        hipMemcpy(cdMpls, mplsArray, (mplsSize * 2 + 1) * sizeof(int), 
                hipMemcpyHostToDevice);

        dim3 blockDim(16,16,1);
        dim3 gridDim(
                (image->width + blockDim.x - 1) / blockDim.x,
                (image->height + blockDim.y - 1) / blockDim.y);
        kernelSetNewVelocities<<<gridDim, blockDim>>>();
        hipDeviceSynchronize();
    }
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  
void
CudaRenderer::clearImage() {
    dim3 blockDim(16,16,1);
    dim3 gridDim(
            (image->width + blockDim.x - 1) / blockDim.x,
            (image->height + blockDim.y - 1) / blockDim.y);
    kernelClearImage<<<gridDim, blockDim>>>(1.f,1.f,0.f,1.f);
    hipDeviceSynchronize();
}

void
CudaRenderer::render() {
//    usleep(1000000);
    dim3 blockDim(16,16,1);
    dim3 gridDim(
            (image->width + blockDim.x - 1) / blockDim.x,
            (image->height + blockDim.y - 1) / blockDim.y);
    
    kernelAdvectVelocityForward<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelAdvectVelocityBackward<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelApplyVorticity<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelApplyVorticityForce<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelApplyDivergence<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelPressureSolve<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelPressureGradient<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
   
    //TO DO: DRAW STUFF
  
    kernelAdvectColorForward<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelAdvectColorBackward<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();


    //advectVelocityForward();
    //advectVelocityBackward();
    //applyVorticity();
    //applyVorticityForce();
    //applyDivergence();
    //pressureSolve();
    //pressureGradient();

    /*// Draw
    for (int i = 0; i < 4*image->width*image->height; i+=4) {
            int grid_row = ((i/4) / image->width) / (CELL_DIM);
            int grid_col = ((i/4) % image->width) / (CELL_DIM);
            double vx = cdVX[grid_row][grid_col];
            double vy = cdVY[grid_row][grid_col];
            double v = sqrt(vx * vx + vy * vy);

            if (abs(v) < 0.00001) {
                // make the cdColor go away faster
                cdColor[grid_row][grid_col][0] *= 0.9;
                cdColor[grid_row][grid_col][1] *= 0.9;
                cdColor[grid_row][grid_col][2] *= 0.9;
                cdColor[grid_row][grid_col][3] = 1.0;
            } 
            cdColor[grid_row][grid_col][0] *= 0.9494; 
            cdColor[grid_row][grid_col][1] *= 0.9494; 
            cdColor[grid_row][grid_col][2] *= 0.9696; 
        
            if (mousePressedLocations.size() > 0) {
                //double d = sqrt(vx * vx + vy * vy);
                double projection;
                std::pair<double,double> mouseSegmentVelocity;
                double l = distanceToNearestMouseSegment(grid_col, grid_row, 
                        &projection, &mouseSegmentVelocity);

                //if (l < 1.0) printf("l is %f\n", l);
                float taperFactor = 0.6;
                double projectedFraction = 1.0 - std::min(1.0, 
                        std::max(projection, 0.0)) * taperFactor;
                double R = 12; //0.025; // the bigger, the more stuff gets cdColored
                double m = exp(-l/R); //drag coefficient
                //double speed = d;
                double vx = cdVX[grid_row][grid_col];
                double vy = cdVY[grid_row][grid_col];
                double speed = sqrt(vx * vx + vy * vy);

                //printf("l is %f, m is %f, projection is %f\n", l, m, projection);

                double x = std::min(1.0, std::max(fabs((speed * speed * 0.02 - 
                            projection * 5.0) * projectedFraction), 0.0));

                double r = (2.4 / 60.0) * x + (0.2 /30.0) * (1-x) + (1.0 * pow(x, 9.0));
                double g = (0.0 / 60.0) * x + (51.8 / 30.0) * (1-x) + (1.0 * pow(x, 9.0));
                double b = (5.9 / 60.0) * x + (100.0 / 30.0) * (1-x) + (1.0 * pow(x, 9.0));

                cdColor[grid_row][grid_col][0] += m * r;
                cdColor[grid_row][grid_col][1] += m * g;
                cdColor[grid_row][grid_col][2] += m * b;
                cdColor[grid_row][grid_col][3] = 1.0;
            }

            image->data[i] = cdColor[grid_row][grid_col][0];
            image->data[i+1] = cdColor[grid_row][grid_col][1];
            image->data[i+2] = cdColor[grid_row][grid_col][2];
            image->data[i+3] = cdColor[grid_row][grid_col][3];
    }
    advectColor();*/
}

