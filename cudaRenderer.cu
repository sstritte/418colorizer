#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <float.h>
#include <utility>
#include <stdio.h>
#include <cstring>
#include <vector>
#include <unistd.h>
#include "cudaRenderer.h"
#include "image.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#define CELL_DIM 1
#define TIME_STEP 1


///////////////////////////CUDA CODE BELOW////////////////////////////////
struct GlobalConstants {
    int cells_per_side;
    int width;
    int height;

    float* VX;
    float* VY;
    float* pressures;
    float* VXCopy;
    float* VYCopy;
    float* divergence;
    float* vorticity;
    float* color;
    float* colorCopy;
    float* imageData;

    int* mpls;
};

__constant__ GlobalConstants cuParams;

// kernelClearImage
__global__ void kernelClearImage(float r, float g, float b, float a) {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuParams.width;
    int height = cuParams.height;

    if (imageX >= width || imageY >= height) return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r,g,b,a);
    
    // Write to global memory.
    *(float4*)(&cuParams.imageData[offset]) = value;

}


__device__ __inline__ int
isBoundary(int i, int j) {
    int cells_per_side = cuParams.cells_per_side;
    if (j == 0) return 1; // left 
    if (i == 0) return 2; // top
    if (j == cells_per_side) return 3; // right
    if (i == cells_per_side) return 4; // bottom
    return 0;
}

// a is prev mouse point, b is cur mouse point, p is point to consider,
// fp is fraction projection to be populated as output
__device__ __inline__ double 
distanceToSegment(double ax, double ay, double bx, double by, 
        double px, double py, double* fp) {
    double dx = px - ax; //vec2 d = p - a;
    double dy = py - ay;
    double xx = bx - ax; //vec2 x = b - a;
    double xy = by - ay;
    *fp = 0.0; // fractional projection, 0 - 1 in the length of b-a
    double lx = sqrt(xx*xx + xy*xy); //length(x)
    double ld = sqrt(dx*dx + dy*dy); //length(d)
    if (lx <= 0.0001) return ld;
    double projection = dx*(xx/lx) + dy*(xy/lx); //dot(d, x/lx)
    *fp = projection / lx;
    if (projection < 0.0) return ld;
    else if (projection > lx) return sqrt((px-bx) * (px-bx) +
            (py-by) * (py-by)); //length(p - b)
    return sqrt(abs(dx*dx + dy*dy - projection * projection));
}

__device__ __inline__ double 
distanceToNearestMouseSegment(double px, double py, double *fp,
        double* vx, double *vy) {
    double minLen = DBL_MAX;
    double fpResult = 0.0;
    double vxResult = 0.0;
    double vyResult = 0.0;
    for (int i = 0; i < 400 - 2; i += 2) {

        int grid_col1 = cuParams.mpls[i];
        int grid_row1 = cuParams.mpls[i + 1];
        int grid_col2 = cuParams.mpls[i + 2];
        int grid_row2 = cuParams.mpls[i + 3];
        if (grid_col2 == 0 & grid_row2 == 0) break;
        double len = distanceToSegment(grid_col1, grid_row1, grid_col2, grid_row2, px, py, fp);
        if (len < minLen) {
            minLen = len;
            fpResult = *fp;
            vxResult = grid_col2 - grid_col1;
            vyResult = grid_row2 - grid_row1;
        }        

    }
    *fp = fpResult;
    *vx = vxResult;
    *vy = vyResult;
    return minLen;
}

//kernelFadeVelocities
__global__ void kernelFadeVelocities() {
    int grid_col = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;
    int height = cuParams.height;

    if (grid_col >= width || grid_row >= height) return;
    if (grid_row * width + grid_col >= width * height) return; 
    
    cuParams.VX[grid_row * width + grid_col] *= 0.999;
    cuParams.VY[grid_row * width + grid_col] *= 0.999;
}

//kernelSetNewVelocities
__global__ void kernelSetNewVelocities() {
    int grid_col = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;
    int height = cuParams.height;

    if (grid_col >= width || grid_row >= height) return;
    if (grid_row * width + grid_col >= width * height) return; 
    
    int imageX = grid_col;
    int imageY = grid_row;
    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(1.f,0.f,1.f,1.f);

    // Write to global memory.
    *(float4*)(&cuParams.imageData[offset]) = value;
   
    cuParams.VX[grid_row * width + grid_col] *= 0.999;
    cuParams.VY[grid_row * width + grid_col] *= 0.999;
    double projection;
    double vx;
    double vy;
    double l = distanceToNearestMouseSegment(grid_col, grid_row, 
            &projection, &vx, &vy);
    //printf("velocity %f,%f\n", mouseSegmentVelocity.first, mouseSegmentVelocity.second);
    double taperFactor = 0.6;
    double projectedFraction = 1.0 - fminf(1.0, fmaxf(projection, 0.0)) * taperFactor;
    double R = 10;
    double m = exp(-l/R); //drag coefficient
    m *= projectedFraction * projectedFraction;
    double targetVelocityX = vx * 1 * 1.4; 
    double targetVelocityY = vy * 1 * 1.4; 

    cuParams.VX[grid_row * width + grid_col] += 
        (targetVelocityX - cuParams.VX[grid_row * width + grid_col]) * m;
    cuParams.VY[grid_row * width + grid_col] += 
        (targetVelocityY - cuParams.VY[grid_row * width + grid_col]) * m;

}

//kernelAdvectVelocityForward
__global__ void kernelAdvectVelocityForward() {
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

    cuParams.VXCopy[row * width + col] = cuParams.VX[row * width + col];
    cuParams.VYCopy[row * width + col] = cuParams.VY[row * width + col];

   int pixelRow = row * CELL_DIM;
   int pixelCol = col * CELL_DIM;
   int nextPixelRow = round(pixelRow + TIME_STEP * cuParams.VY[row * width + col] * CELL_DIM);
   int nextPixelCol = round(pixelCol + TIME_STEP * cuParams.VX[row * width + col] * CELL_DIM);
   int nextCellCol = nextPixelCol / CELL_DIM;
   int nextCellRow = nextPixelRow / CELL_DIM;

   if (nextCellCol < cells_per_side && nextCellRow < cells_per_side 
           && nextCellCol >= 0 && nextCellRow >= 0) {
        cuParams.VX[nextCellRow * width + nextCellCol] = cuParams.VXCopy[row * width + col];
        cuParams.VY[nextCellRow * width + nextCellCol] = cuParams.VYCopy[row* width + col];
   } 

}

//kernelAdvectVelocityBackward
__global__ void kernelAdvectVelocityBackward() {
    int cells_per_side = cuParams.cells_per_side;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int width = cuParams.width;

   int pixelRow = row * CELL_DIM;
   int pixelCol = col * CELL_DIM;
   int prevPixelRow = round(pixelRow - TIME_STEP * cuParams.VY[row * width + col] * CELL_DIM);
   int prevPixelCol = round(pixelCol - TIME_STEP * cuParams.VX[row * width + col] * CELL_DIM);
   int prevCellCol = prevPixelCol / CELL_DIM;
   int prevCellRow = prevPixelRow / CELL_DIM;

   if (prevCellCol < cells_per_side && prevCellRow < cells_per_side 
           && prevCellCol >= 0 && prevCellRow >= 0) {
        cuParams.VX[row * width + col] = cuParams.VXCopy[prevCellRow * width + prevCellCol];
        cuParams.VY[row * width + col] = cuParams.VYCopy[prevCellRow * width + prevCellCol];
   } 
   if (prevCellCol == col && prevCellRow == row) {
        // you don't move so just disappear
        cuParams.VX[row * width + col] = 0;
        cuParams.VY[row * width + col] = 0;
   }
}


//////////////////////////////////////////////////////////////////////////
///////////////////////////HOST CODE BELOW////////////////////////////////
//////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
    image = NULL;

    VX = NULL;
    VY = NULL;
    color = NULL;
    colorCopy = NULL;
    pressures = NULL;
    VXCopy = NULL;
    VYCopy = NULL;
    divergence = NULL;
    vorticity = NULL;

    mpls = NULL;

    cdVX = NULL;
    cdVY = NULL;
    cdColor = NULL;
    cdColorCopy = NULL;
    cdPressures = NULL;
    cdVXCopy = NULL;
    cdVYCopy = NULL;
    cdDivergence = NULL;
    cdVorticity = NULL;
    cdImageData = NULL;

    cdMpls = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) delete image;

    if (VX) {
        delete VX;
        delete VY;
        delete pressures;
        delete VXCopy;
        delete VYCopy;
        delete divergence;
        delete vorticity;
        delete color;
        delete colorCopy;
        delete mpls;
    }

    if (cdVX) {
        hipFree(cdVX);
        hipFree(cdVY);
        hipFree(cdPressures);
        hipFree(cdVXCopy);
        hipFree(cdVYCopy);
        hipFree(cdDivergence);
        hipFree(cdVorticity);
        hipFree(cdColor);
        hipFree(cdColorCopy);
        hipFree(cdImageData);
        hipFree(cdMpls);
    }
}

const Image*
CudaRenderer::getImage() {
    printf("Copying image data from device\n");

    hipMemcpy(image->data, cdImageData, 
            4 * sizeof(float) * image->width * image->height,
            hipMemcpyDeviceToHost);

    return image;
}


void
CudaRenderer::setup() {
   cells_per_side = image->width / CELL_DIM - 1;

   hipMalloc(&cdVX, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdVY, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   //hipMalloc(&cdPressures, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdVXCopy, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdVYCopy, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   /*hipMalloc(&cdDivergence, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdVorticity, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdColor, 4 * sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMalloc(&cdColorCopy, 4 * sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));*/
   hipMalloc(&cdImageData, 4 * sizeof(float) * image->width * image->height);
   hipMalloc(&cdMpls, 400 * sizeof(float) * image->width * image->height);

   hipMemset(cdVX, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdVY, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   //hipMemset(cdPressures, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdVXCopy, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdVYCopy, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   /*hipMemset(cdDivergence, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdVorticity, 0, sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdColor, 0, 4 * sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));
   hipMemset(cdColorCopy, 0, 4 * sizeof(float) * (cells_per_side + 1) * (cells_per_side + 1));*/

    GlobalConstants params;
    params.cells_per_side = cells_per_side;
    params.width = image->width;
    params.height = image->height;
    params.VX = cdVX;
    params.VY = cdVY;
    //params.pressures = cdPressures;
    params.VXCopy = cdVXCopy;
    params.VYCopy = cdVYCopy;
    /*params.divergence = cdDivergence;
    params.vorticity = cdVorticity;
    params.color = cdColor;
    params.colorCopy = cdColorCopy;*/
    params.imageData = cdImageData;
    params.mpls = cdMpls;

    hipMemcpyToSymbol(HIP_SYMBOL(cuParams), &params, sizeof(GlobalConstants));
}

// Called after clear, before render
void CudaRenderer::setNewQuantities(std::vector<std::pair<int, int> > mpls) {

    int mplsSize = mpls.size();
    if (mplsSize == 0) {
        // if mpls.size is 0, then call kernel that decreases VX,VY by 0.999
        dim3 blockDim(16,16,1);
        dim3 gridDim(
                (image->width + blockDim.x - 1) / blockDim.x,
                (image->height + blockDim.y - 1) / blockDim.y);
        kernelFadeVelocities<<<gridDim, blockDim>>>();
        hipDeviceSynchronize();

    } else {
        int* mplsArray = new int[mplsSize * 2 + 1];
        int count = 0;
        for (std::vector<std::pair<int,int> >::iterator it = mpls.begin() 
                ; it != mpls.end(); ++it) {
            std::pair<int,int> c = *it;
            mplsArray[count] = c.first;
            mplsArray[count + 1] = c.second;
            count += 2;
        }
        hipMemset(cdMpls, 0, 400 * sizeof(int));
        hipMemcpy(cdMpls, mplsArray, (mplsSize * 2 + 1) * sizeof(int), 
                hipMemcpyHostToDevice);

        dim3 blockDim(16,16,1);
        dim3 gridDim(
                (image->width + blockDim.x - 1) / blockDim.x,
                (image->height + blockDim.y - 1) / blockDim.y);
        kernelSetNewVelocities<<<gridDim, blockDim>>>();
        hipDeviceSynchronize();
    }
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  
void
CudaRenderer::clearImage() {
    dim3 blockDim(16,16,1);
    dim3 gridDim(
            (image->width + blockDim.x - 1) / blockDim.x,
            (image->height + blockDim.y - 1) / blockDim.y);
    kernelClearImage<<<gridDim, blockDim>>>(1.f,1.f,0.f,1.f);
    hipDeviceSynchronize();
}

/*void CudaRenderer::advectColorBackward() {
     //Advecting the values in cdColor
     for (int row = 0; row < cells_per_side; row++) {
         for (int col = 0; col < cells_per_side; col++) {
            int pixelRow = row * CELL_DIM;// + CELL_DIM/2;
            int pixelCol = col * CELL_DIM;// + CELL_DIM/2;
            int prevPixelRow = round(pixelRow - TIME_STEP * cdVY[row][col] * CELL_DIM);
            int prevPixelCol = round(pixelCol - TIME_STEP * cdVX[row][col] * CELL_DIM);
            int prevCellCol = prevPixelCol / CELL_DIM;
            int prevCellRow = prevPixelRow / CELL_DIM;
 
            if (prevCellCol < cells_per_side && prevCellRow < cells_per_side 
                    && prevCellCol >= 0 && prevCellRow >= 0) {
                 cdColor[row][col][0] = cdColorCopy[prevCellRow][prevCellCol][0];
                 cdColor[row][col][1] = cdColorCopy[prevCellRow][prevCellCol][1];
                 cdColor[row][col][2] = cdColorCopy[prevCellRow][prevCellCol][2];
                 cdColor[row][col][3] = cdColorCopy[prevCellRow][prevCellCol][3];
            } 
         }
     }
 }
 
 void CudaRenderer::advectColorForward() {
     //Advecting the values in cdColor
     for (int row = 0; row < cells_per_side; row++) {
         for (int col = 0; col < cells_per_side; col++) {
            int pixelRow = row * CELL_DIM;// + CELL_DIM/2;
            int pixelCol = col * CELL_DIM;// + CELL_DIM/2;
            int nextPixelRow = round(pixelRow + TIME_STEP * cdVY[row][col] * CELL_DIM);
            int nextPixelCol = round(pixelCol + TIME_STEP * cdVX[row][col] * CELL_DIM);
            int nextCellCol = nextPixelCol / CELL_DIM;
            int nextCellRow = nextPixelRow / CELL_DIM;
 
            if (nextCellCol < cells_per_side && nextCellRow < cells_per_side 
                    && nextCellCol >= 0 && nextCellRow >= 0) {
                 cdColor[nextCellRow][nextCellCol][0] = cdColorCopy[row][col][0];
                 cdColor[nextCellRow][nextCellCol][1] = cdColorCopy[row][col][1];
                 cdColor[nextCellRow][nextCellCol][2] = cdColorCopy[row][col][2];
                 cdColor[nextCellRow][nextCellCol][3] = cdColorCopy[row][col][3];
            } 
         }
     }
 }
 

void
CudaRenderer::advectColor() {
    for (int i = 0; i < cells_per_side + 1; i++) {
        for (int j = 0; j < cells_per_side + 1; j++) {
            for (int k = 0; k < 4; k++) {
                cdColorCopy[i][j][k] = cdColor[i][j][k];
            }
        }
    }
    advectColorForward();
    advectColorBackward();
}


void
CudaRenderer::applyPressure() {
    for (int i = 0; i < cells_per_side; i ++) {
        for (int j = 0; j < cells_per_side; j++) {
            float force_x = (cdPressures[i][j] - cdPressures[i][j+1]);
            float force_y = (cdPressures[i][j] - cdPressures[i+1][j]);
            cdVX[i][j] += force_x;
            cdVX[i][j+1] += force_x;
            cdVY[i][j] += force_y;
            cdVY[i+1][j] += force_y;
            //if (force_x != 0) printf("adding force_x %f to (%d,%d)\n", force_x, i, j);
        }
    }
}


// Divergence of velocity: This computes how divergent the velocity field is
// (how much in/out flow there is at every point).  Used as input to the 
// pressure solve below.
void
CudaRenderer::applyDivergence() {
    float L = 0.0;
    float R = 0.0;
    float B = 0.0;
    float T = 0.0;
    for (int i = 0; i < cells_per_side + 1; i++) {
        for (int j = 0; j < cells_per_side + 1; j++) {
            if (isBoundary(i,j)) continue;
            
            if (i > 0) T = cdVY[i-1][j];
            if (i < cells_per_side) B = cdVY[i+1][j];
            if (j < cells_per_side) R = cdVX[i][j+1];
            if (j > 0) L = cdVX[i][j-1];
            cdDivergence[i][j] = 0.5*((R-L) + (T-B));   
        }
    }
}

void
CudaRenderer::pressureSolve() {
    float L = 0.0;
    float R = 0.0;
    float B = 0.0;
    float T = 0.0;
    float** tempPressure = new float*[cells_per_side + 1];
    for(int i = 0; i < cells_per_side+1; i++) {
        tempPressure[i] = new float[cells_per_side+1];
    }

    for (int i = 0; i < cells_per_side + 1; i++) {
        for (int j = 0; j < cells_per_side + 1; j++) {
            if (isBoundary(i,j)) continue;
            if (i > 0) T = cdPressures[i-1][j];
            if (i < cells_per_side) B = cdPressures[i+1][j];
            if (j < cells_per_side) R = cdPressures[i][j+1];
            if (j > 0) L = cdPressures[i][j-1];
            tempPressure[i][j] = (L + R + B + T + -1 * cdDivergence[i][j]) * .25;
                //if (L+R+B+T > 0.0) printf("L+R+B+T is %f, -1*cdDivergence is %f\n", L+R+B+T, -1*cdDivergence[i][j]);
        }
    }
    for (int i = 0; i < cells_per_side + 1; i++) {
        for (int j = 0; j < cells_per_side + 1; j++) { 
            if (isBoundary(i,j)) continue;
            cdPressures[i][j] = tempPressure[i][j];
        }
    }

}

void
CudaRenderer::pressureGradient() {
    float L = 0.0;
    float R = 0.0;
    float B = 0.0;
    float T = 0.0;
    for (int i = 0; i < cells_per_side + 1; i++) {
        for (int j = 0; j < cells_per_side + 1; j++) {
            if (isBoundary(i,j)) continue;
            if (i > 0) T = cdPressures[i-1][j];
            if (i < cells_per_side) B = cdPressures[i+1][j];
            if (j < cells_per_side) R = cdPressures[i][j+1];
            if (j > 0) L = cdPressures[i][j-1];

            //if (cdVY[i][j] > 10) printf("doing cdVY = %f - 0.5*(%f)\n", cdVY[i][j], T-B);
            cdVX[i][j] = cdVX[i][j] - 0.5*(R - L);
            cdVY[i][j] = cdVY[i][j] - 0.5*(T - B);
            //if (cdVY[i][j] != 0.0) printf("cdVY is %f\n", cdVY[i][j]);
        }
    }
}

void
CudaRenderer::applyVorticity() {
    float L = 0.0;
    float R = 0.0;
    float B = 0.0;
    float T = 0.0;
    for (int i = 0; i < cells_per_side + 1; i++) {
        for (int j = 0; j < cells_per_side + 1; j++) {
            if (isBoundary(i,j)) continue;
            if (i > 0) T = cdVX[i-1][j];
            if (i < cells_per_side) B = cdVX[i+1][j];
            if (j < cells_per_side) R = cdVY[i][j+1];
            if (j > 0) L = cdVY[i][j-1];
            cdVorticity[i][j] = 0.5 * ((R - L) - (T - B));
        }
    }
}

void
CudaRenderer::applyVorticityForce() {
    float vortConfinementFloat = 0.035f;
    float vortL = 0.0;
    float vortR = 0.0;
    float vortB = 0.0;
    float vortT = 0.0;
    float vortC = 0.0;
    for (int i = 0; i < cells_per_side + 1; i++) {
        for (int j = 0; j < cells_per_side + 1; j++) {
            if (isBoundary(i,j)) continue;
            if (i > 0) vortT = cdVorticity[i-1][j];
            if (i < cells_per_side) vortB = cdVorticity[i+1][j];
            if (j < cells_per_side) vortR = cdVorticity[i][j+1];
            if (j > 0) vortL = cdVorticity[i][j-1];
            vortC = cdVorticity[i][j];
            float forceX = 0.5 * (abs(vortT) - abs(vortB));
            float forceY = 0.5 * (abs(vortR) - abs(vortL));
            float EPSILON = pow(2,-12);
            float magSqr = std::max(EPSILON, forceX * forceX + forceY * forceY);
            forceX = forceX * (1/sqrt(magSqr));
            forceY = forceY * (1/sqrt(magSqr));
            forceX *= vortConfinementFloat * vortC * 1;
            forceY *= vortConfinementFloat * vortC * -1;
            cdVX[i][j] += forceX;
            cdVY[i][j] += forceY;
        }
    }
}
*/
void
CudaRenderer::render() {
//    usleep(1000000);
    dim3 blockDim(16,16,1);
    dim3 gridDim(
            (image->width + blockDim.x - 1) / blockDim.x,
            (image->height + blockDim.y - 1) / blockDim.y);
    kernelAdvectVelocityForward<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    kernelAdvectVelocityBackward<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();


/*    advectVelocityForward();
    advectVelocityBackward();
    applyVorticity();
    applyVorticityForce();
    applyDivergence();
    pressureSolve();
    pressureGradient();

    // Draw
    for (int i = 0; i < 4*image->width*image->height; i+=4) {
            int grid_row = ((i/4) / image->width) / (CELL_DIM);
            int grid_col = ((i/4) % image->width) / (CELL_DIM);
            double vx = cdVX[grid_row][grid_col];
            double vy = cdVY[grid_row][grid_col];
            double v = sqrt(vx * vx + vy * vy);

            if (abs(v) < 0.00001) {
                // make the cdColor go away faster
                cdColor[grid_row][grid_col][0] *= 0.9;
                cdColor[grid_row][grid_col][1] *= 0.9;
                cdColor[grid_row][grid_col][2] *= 0.9;
                cdColor[grid_row][grid_col][3] = 1.0;
            } 
            cdColor[grid_row][grid_col][0] *= 0.9494; 
            cdColor[grid_row][grid_col][1] *= 0.9494; 
            cdColor[grid_row][grid_col][2] *= 0.9696; 
        
            if (mousePressedLocations.size() > 0) {
                //double d = sqrt(vx * vx + vy * vy);
                double projection;
                std::pair<double,double> mouseSegmentVelocity;
                double l = distanceToNearestMouseSegment(grid_col, grid_row, 
                        &projection, &mouseSegmentVelocity);

                //if (l < 1.0) printf("l is %f\n", l);
                float taperFactor = 0.6;
                double projectedFraction = 1.0 - std::min(1.0, 
                        std::max(projection, 0.0)) * taperFactor;
                double R = 12; //0.025; // the bigger, the more stuff gets cdColored
                double m = exp(-l/R); //drag coefficient
                //double speed = d;
                double vx = cdVX[grid_row][grid_col];
                double vy = cdVY[grid_row][grid_col];
                double speed = sqrt(vx * vx + vy * vy);

                //printf("l is %f, m is %f, projection is %f\n", l, m, projection);

                double x = std::min(1.0, std::max(fabs((speed * speed * 0.02 - 
                            projection * 5.0) * projectedFraction), 0.0));

                double r = (2.4 / 60.0) * x + (0.2 /30.0) * (1-x) + (1.0 * pow(x, 9.0));
                double g = (0.0 / 60.0) * x + (51.8 / 30.0) * (1-x) + (1.0 * pow(x, 9.0));
                double b = (5.9 / 60.0) * x + (100.0 / 30.0) * (1-x) + (1.0 * pow(x, 9.0));

                cdColor[grid_row][grid_col][0] += m * r;
                cdColor[grid_row][grid_col][1] += m * g;
                cdColor[grid_row][grid_col][2] += m * b;
                cdColor[grid_row][grid_col][3] = 1.0;
            }

            image->data[i] = cdColor[grid_row][grid_col][0];
            image->data[i+1] = cdColor[grid_row][grid_col][1];
            image->data[i+2] = cdColor[grid_row][grid_col][2];
            image->data[i+3] = cdColor[grid_row][grid_col][3];
    }
    advectColor();*/
}

