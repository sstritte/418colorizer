#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "util.h"
#include "cycleTimer.h"

/*
 * THIS IS BASICALLY THE SAME AS refRenderer RIGHT NOW
 *
 */

CudaRenderer::CudaRenderer() {
    image = NULL;
    mousePressedLocation = NULL; 
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

}

const Image*
CudaRenderer::getImage() {
    return image;
}

void
CudaRenderer::setup() {
    // Nothing for now because not actually using CUDA yet
}

void 
CudaRenderer::setMousePressedLocation(int* mpl) {
    mousePressedLocation = mpl;
}
 
void 
CudaRenderer::setNewQuantities(double* vxs, double* vys, double* ps) {
    // nothing yet
}
// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {
    image->clear(1.f,1.f,1.f,1.f);
}

void
CudaRenderer::render() {
    for (int i = 0; i < 4*image->width*image->height; i+=4) {
        if (mousePressedLocation[i / 4]) {
            image->data[i] = 1.0;
            image->data[i+1] = 1.0;
            image->data[i+2] = 1.0;
            image->data[i+3] = 1.0;
        } else {
            image->data[i] = 0.1798;
            image->data[i+1] = 0.457;
            image->data[i+2] = 0.9063;
            image->data[i+3] = 0.5;
        }
    }
}
